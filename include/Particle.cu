#include "hip/hip_runtime.h"
#ifndef PFILTER_PARTICLE
#define PFILTER_PARTICLE

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <eigen3/Eigen/Core>

__global__ template <int N>
class Particle
{
public:
  Eigen::Matrix<float, N, 1> getState();
  Eigen::Matrix<float, N, N> getCovariance();

  __host__ __device__ int getDims();
  __host__ __device__ int getState();

private:
  Eigen::Matrix<float, N, 1> m_state;
  Eigen::Matrix<float, N, N> m_cov;
  static const int m_dims = N;
};

#endif