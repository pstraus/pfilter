#include "hip/hip_runtime.h"

#pragma once

#include <pfilter/include/I_Filter.h>
#include "Particle.cu"

__global__ class ConicParticleFilter : public I_2DFilter
{
public:
  __host__ __device__ ConicParticleFilter(uint numParticles);
  __host__ __device__ ~ConicParticleFilter() override = default;
  __host__ Eigen::Matrix<double, 4, 1> getStateEstimate() const override;
  __host__ Eigen::Matrix<double, 4, 4> getCovarianceEstimate() const override;

  __host__ void updateFilter(const Eigen::Matrix<float, 1, 1> &data) override;

private:
  uint mNumParticles;
  Particle[] mp_particleArray;
};

void updateFilter(int n, double *track, double *data)
{
}
